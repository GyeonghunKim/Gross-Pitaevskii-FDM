#include "hip/hip_runtime.h"
/**
 * @file cn_rect_psolver.cu
 * @author Gyeonghun Kim, Minyoung Kim
 * @brief Implementation file for CUDA based parallel crank nicolson solver
 * @version 0.1
 * @date 2022-06-05
 * 
 * @copyright Copyright (c) 2022
 * 
 */
#define NVTX_USE true
#include "cn_rect_psolver.cuh"

__global__ void cn_rect_cusolver(float *psi_old_real,
                                 float *psi_old_imag,
                                 float *psi_new_real_trial,
                                 float *psi_new_imag_trial,
                                 float *psi_new_real,
                                 float *psi_new_imag,
                                 float *potential,
                                 int n_x,
                                 int n_y,
                                 float g,
                                 float h_x,
                                 float h_y,
                                 float tau,
                                 float relaxation)
{
    __shared__ float tile_old_real[nTx][nTy];
    __shared__ float tile_old_imag[nTx][nTy];
    __shared__ float tile_new_real_trial[nTx][nTy];
    __shared__ float tile_new_imag_trial[nTx][nTy];
    __shared__ float tile_potential[nTx][nTy];

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int i = (block_x + thread_x);
    int j = (block_y + thread_y);
    int striding = gridDim.x * blockDim.x;

    tile_old_real[thread_x][thread_y] = psi_old_real[j * striding + i];
    tile_old_imag[thread_x][thread_y] = psi_old_imag[j * striding + i];
    tile_new_real_trial[thread_x][thread_y] = psi_new_real_trial[j * striding + i];
    tile_new_imag_trial[thread_x][thread_y] = psi_new_imag_trial[j * striding + i];
    tile_potential[thread_x][thread_y] = potential[j * striding + i];

    __syncthreads();

    // Parameters
    float sigma_x = tau / (4 * h_x * h_x);
    float sigma_y = tau / (4 * h_y * h_y);
    float a = 2 * sigma_x + 2 * sigma_y + 0.5 * tau * tile_potential[thread_x][thread_y];
    float b = a * a + 1;
    float amplitude_old = tile_old_real[thread_x][thread_y] * tile_old_real[thread_x][thread_y] + tile_old_imag[thread_x][thread_y] * tile_old_imag[thread_x][thread_y];
    float amplitude_new = tile_new_real_trial[thread_x][thread_y] * tile_new_real_trial[thread_x][thread_y] + tile_new_imag_trial[thread_x][thread_y] * tile_new_imag_trial[thread_x][thread_y];

    // Update tile position
    atomicAdd(&psi_new_real[j * striding + i],
              relaxation * ((1 - a * a) / b * tile_old_real[thread_x][thread_y] + 2 * a / b * tile_old_imag[thread_x][thread_y]) * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_real[j * striding + i],
              relaxation * (-a * g * tau / (2 * b) * tile_old_real[thread_x][thread_y] + g * tau / (2 * b) * tile_old_imag[thread_x][thread_y]) * amplitude_old * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_real[j * striding + i],
              relaxation * (-a * g * tau / (2 * b) * tile_new_real_trial[thread_x][thread_y] + g * tau / (2 * b) * tile_new_imag_trial[thread_x][thread_y]) * amplitude_new * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y));

    atomicAdd(&psi_new_imag[j * striding + i],
              relaxation * (-2 * a / b * tile_old_real[thread_x][thread_y] + (1 - a * a) / b * tile_old_imag[thread_x][thread_y]) * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_imag[j * striding + i],
              relaxation * (-g * tau / (2 * b) * tile_old_real[thread_x][thread_y] - a * g * tau / (2 * b) * tile_old_imag[thread_x][thread_y]) * amplitude_old * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_imag[j * striding + i],
              relaxation * (-g * tau / (2 * b) * tile_new_real_trial[thread_x][thread_y] - a * g * tau / (2 * b) * tile_new_imag_trial[thread_x][thread_y]) * amplitude_new * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y));

    // Update left
    atomicAdd(&psi_new_real[j * striding + i - 1],
              relaxation * (a * sigma_x / b * tile_old_real[thread_x][thread_y] - sigma_x / b * tile_old_imag[thread_x][thread_y]) *
                  (i > 0) * (i < n_x) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_real[j * striding + i - 1],
              relaxation * (a * sigma_x / b * tile_new_real_trial[thread_x][thread_y] - sigma_x / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i > 0) * (i < n_x) * (j >= 0) * (j < n_y));

    atomicAdd(&psi_new_imag[j * striding + i - 1],
              relaxation * (sigma_x / b * tile_old_real[thread_x][thread_y] + a * sigma_x / b * tile_old_imag[thread_x][thread_y]) *
                  (i > 0) * (i < n_x) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_imag[j * striding + i - 1],
              relaxation * (sigma_x / b * tile_new_real_trial[thread_x][thread_y] + a * sigma_x / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i > 0) * (i < n_x) * (j >= 0) * (j < n_y));

    // Update right
    atomicAdd(&psi_new_real[j * striding + i + 1],
              relaxation * (a * sigma_x / b * tile_old_real[thread_x][thread_y] - sigma_x / b * tile_old_imag[thread_x][thread_y]) *
                  (i >= 0) * (i < (n_x - 1)) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_real[j * striding + i + 1],
              relaxation * (a * sigma_x / b * tile_new_real_trial[thread_x][thread_y] - sigma_x / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i >= 0) * (i < (n_x - 1)) * (j >= 0) * (j < n_y));

    atomicAdd(&psi_new_imag[j * striding + i + 1],
              relaxation * (sigma_x / b * tile_old_real[thread_x][thread_y] + a * sigma_x / b * tile_old_imag[thread_x][thread_y]) *
                  (i >= 0) * (i < (n_x - 1)) * (j >= 0) * (j < n_y));
    atomicAdd(&psi_new_imag[j * striding + i + 1],
              relaxation * (sigma_x / b * tile_new_real_trial[thread_x][thread_y] + a * sigma_x / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i >= 0) * (i < (n_x - 1)) * (j >= 0) * (j < n_y));

    // Update down
    atomicAdd(&psi_new_real[(j - 1) * striding + i],
              relaxation * (a * sigma_y / b * tile_old_real[thread_x][thread_y] - sigma_y / b * tile_old_imag[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j > 0) * (j < n_y));
    atomicAdd(&psi_new_real[(j - 1) * striding + i],
              relaxation * (a * sigma_y / b * tile_new_real_trial[thread_x][thread_y] - sigma_y / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j > 0) * (j < n_y));

    atomicAdd(&psi_new_imag[(j - 1) * striding + i],
              relaxation * (sigma_y / b * tile_old_real[thread_x][thread_y] + a * sigma_y / b * tile_old_imag[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j > 0) * (j < n_y));
    atomicAdd(&psi_new_imag[(j - 1) * striding + i],
              relaxation * (sigma_y / b * tile_new_real_trial[thread_x][thread_y] + a * sigma_y / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j > 0) * (j < n_y));

    // Update up
    atomicAdd(&psi_new_real[(j + 1) * striding + i],
              relaxation * (a * sigma_y / b * tile_old_real[thread_x][thread_y] - sigma_y / b * tile_old_imag[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j >= 0) * (j < (n_y - 1)));
    atomicAdd(&psi_new_real[(j + 1) * striding + i],
              relaxation * (a * sigma_y / b * tile_new_real_trial[thread_x][thread_y] - sigma_y / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j >= 0) * (j < (n_y - 1)));

    atomicAdd(&psi_new_imag[(j + 1) * striding + i],
              relaxation * (sigma_y / b * tile_old_real[thread_x][thread_y] + a * sigma_y / b * tile_old_imag[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j >= 0) * (j < (n_y - 1)));
    atomicAdd(&psi_new_imag[(j + 1) * striding + i],
              relaxation * (sigma_y / b * tile_new_real_trial[thread_x][thread_y] + a * sigma_y / b * tile_new_imag_trial[thread_x][thread_y]) *
                  (i >= 0) * (i < n_x) * (j >= 0) * (j < (n_y - 1)));
}
// Only works with single block
__global__ void calculate_local_error(float *psi_1_real,
                                      float *psi_1_imag,
                                      float *psi_2_real,
                                      float *psi_2_imag,
                                      float *error_array,
                                      int n_x,
                                      int n_y)
{

    __shared__ float tile_psi_1_real[nTx][nTy];
    __shared__ float tile_psi_1_imag[nTx][nTy];
    __shared__ float tile_psi_2_real[nTx][nTy];
    __shared__ float tile_psi_2_imag[nTx][nTy];

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int i = (block_x + thread_x);
    int j = (block_y + thread_y);
    int striding = gridDim.x * blockDim.x;

    tile_psi_1_real[thread_x][thread_y] = psi_1_real[j * striding + i];
    tile_psi_1_imag[thread_x][thread_y] = psi_1_imag[j * striding + i];
    tile_psi_2_real[thread_x][thread_y] = psi_2_real[j * striding + i];
    tile_psi_2_imag[thread_x][thread_y] = psi_2_imag[j * striding + i];

    __syncthreads();

    error_array[j * striding + i] = ((tile_psi_1_real[thread_x][thread_y] - tile_psi_2_real[thread_x][thread_y]) * (tile_psi_1_real[thread_x][thread_y] - tile_psi_2_real[thread_x][thread_y])) * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y);
    error_array[j * striding + i] += ((tile_psi_1_imag[thread_x][thread_y] - tile_psi_2_imag[thread_x][thread_y]) * (tile_psi_1_imag[thread_x][thread_y] - tile_psi_2_imag[thread_x][thread_y])) * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y);
}

__global__ void reduction_error(float *error_array, float *error, int array_size)
{

    int idx = threadIdx.x;
    float sum = 0;
    for (int i = idx; i < array_size; i += nTx * nTy)
    {
        sum += error_array[i];
    }

    __shared__ float r[nTx * nTy];
    r[idx] = sum;
    __syncthreads();
    for (int size = nTx * nTy / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *error = r[0];
}

__global__ void calculate_probability(float *psi_real, float *psi_imag, float *probability, int n_x, int n_y)
{

    __shared__ float tile_psi_real[nTx][nTy];
    __shared__ float tile_psi_imag[nTx][nTy];

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int i = (block_x + thread_x);
    int j = (block_y + thread_y);
    int striding = gridDim.x * blockDim.x;

    tile_psi_real[thread_x][thread_y] = psi_real[j * striding + i];
    tile_psi_imag[thread_x][thread_y] = psi_imag[j * striding + i];
    __syncthreads();

    probability[j * striding + i] = ((tile_psi_real[thread_x][thread_y] * tile_psi_real[thread_x][thread_y]) + (tile_psi_imag[thread_x][thread_y] * tile_psi_imag[thread_x][thread_y])) * (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y);
}

__global__ void calculate_normalize_factor(float *probability, float *normalize_factor, int array_size, float unit_area)
{

    int idx = threadIdx.x;
    float sum = 0;
    for (auto i = idx; i < array_size; i += nTx * nTy)
    {
        sum += probability[i];
    }

    __shared__ float r[nTx * nTy];
    r[idx] = sum;
    __syncthreads();

    for (auto size = nTx * nTy / 2; size > 0; size /= 2)
    {
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
    {
        *normalize_factor = sqrt(r[0] * unit_area);
    }
}

__global__ void normalize(float *psi_real, float *psi_imag, float *normalize_factor)
{

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int i = (block_x + thread_x);
    int j = (block_y + thread_y);
    int striding = gridDim.x * blockDim.x;

    psi_real[j * striding + i] /= *normalize_factor;
    psi_imag[j * striding + i] /= *normalize_factor;
}

__global__ void scale_prev_solution(float *psi_real, float *psi_imag, float scale)
{

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int i = (block_x + thread_x);
    int j = (block_y + thread_y);
    int striding = gridDim.x * blockDim.x;

    psi_real[j * striding + i] *= scale;
    psi_imag[j * striding + i] *= scale;
}

CNRectPSolver::CNRectPSolver(
    // std::function<float(float, float)> potential,
    float g,
    RectangularDomain *domain_,
    int device_number)
    : BaseSolver(g)
{
    this->domain = domain_;
    this->string_info = std::string{"Crank_Nicolson_parallel_"};
    hipSetDevice(device_number);
};

void fileout_debug(float *array, int n_x, int n_y, std::string filename)
{
    std::ofstream fileout(filename.data());
    for (auto i = 0; i < n_y; ++i)
    {
        for (auto j = 0; j < n_x - 1; ++j)
        {
            fileout << array[n_x * i + j] << ", ";
        }
        fileout << array[n_x * i + n_x - 1] << std::endl;
    }
}

// void CNRectPSolver::solve(float tolerance, int max_iter)
void CNRectPSolver::solve(float tolerance, int max_iter, std::string dir_name, bool print_info, bool save_data)
{
    hipDeviceSynchronize();
    if (NVTX_USE)
    {
        nvtxRangePushA("solver started");
    }
    int n_x = this->domain->get_num_grid_1();
    int n_y = this->domain->get_num_grid_2();
    float dt = this->domain->get_dt();

    float *d_error;
    float *d_normalize_factor;
    float relaxation_parameter = 1.;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));

    float *d_psi_old_real, *d_psi_old_imag, *d_psi_new_real, *d_psi_new_real_trial, *d_psi_new_imag_trial, *d_psi_new_imag, *d_potential;
    float *d_probability_array, *d_error_array;

    if (NVTX_USE)
    {
        nvtxRangePushA("cuda malloc");
    }
    hipMalloc((float **)&d_psi_old_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_old_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_new_real_trial, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_new_imag_trial, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_potential, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_probability_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error, sizeof(float));
    hipMalloc((float **)&d_normalize_factor, sizeof(float));
    if (NVTX_USE)
    {
        nvtxRangePop();
    }

    hipStream_t stream_psi_new_real, stream_psi_new_imag, stream_potential;
    hipStream_t stream_device_to_device_1, stream_device_to_device_2, stream_device_to_device_3, stream_device_to_device_4;

    hipStreamCreate(&stream_psi_new_real);
    hipStreamCreate(&stream_psi_new_imag);
    hipStreamCreate(&stream_potential);

    hipStreamCreate(&stream_device_to_device_1);
    hipStreamCreate(&stream_device_to_device_2);
    hipStreamCreate(&stream_device_to_device_3);
    hipStreamCreate(&stream_device_to_device_4);

    float *h_psi_new_real, *h_psi_new_imag, *h_potential;
    hipHostAlloc((void **)&h_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipHostMallocDefault);
    hipHostAlloc((void **)&h_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipHostMallocDefault);
    hipHostAlloc((void **)&h_potential, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipHostMallocDefault);

    if (NVTX_USE)
    {
        nvtxRangePushA("initialize: host array");
    }
    std::complex<float> wave_func;
    float potential_value;
    for (int i = 0; i < n_x; ++i)
    {
        for (int j = 0; j < n_y; ++j)
        {
            wave_func = this->domain->at(i, j, 0)->value;
            potential_value = this->domain->potential_grid->at(i, j)->value.real();
            h_psi_new_real[j * TPB.x * nBlocks.x + i] = wave_func.real();
            h_psi_new_imag[j * TPB.x * nBlocks.x + i] = wave_func.imag();
            h_potential[j * TPB.x * nBlocks.x + i] = potential_value;
        }
    }

    if (NVTX_USE)
    {
        nvtxRangePop();
    }

    if (NVTX_USE)
    {
        nvtxRangePushA("memcpyAsync");
    }
    hipMemcpyAsync(d_psi_new_real, h_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice, stream_psi_new_real);
    hipMemcpyAsync(d_psi_new_imag, h_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice, stream_psi_new_imag);
    hipMemcpyAsync(d_potential, h_potential, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice, stream_potential);
    hipMemcpyAsync(d_psi_new_real_trial, d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_1);
    hipMemcpyAsync(d_psi_new_imag_trial, d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_2);
    hipMemcpyAsync(d_psi_old_real, d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_3);
    hipMemcpyAsync(d_psi_old_imag, d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_4);

    if (NVTX_USE)
    {
        nvtxRangePop();
    }

    if (save_data)
    {

        this->domain->generate_directory_name(this->string_info + dir_name, print_info);
        this->domain->generate_single_txt_file(std::string("Solution_") + std::to_string(0), true);
    }
    else
    {
        this->domain->update_time(true);
    }

    float *buffer_real = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *buffer_imag = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);

    std::vector<std::thread> threads;
    threads.reserve(this->domain->get_num_times() - 1);
    for (auto k = 0; k < this->domain->get_num_times() - 1; ++k)
    {
        {
            this->solve_single_time(k, d_psi_old_real,
                                    d_psi_old_imag,
                                    d_psi_new_real_trial,
                                    d_psi_new_imag_trial,
                                    d_psi_new_real,
                                    d_psi_new_imag,
                                    d_potential,
                                    d_probability_array,
                                    d_normalize_factor,
                                    d_error_array,
                                    d_error,
                                    max_iter,
                                    tolerance,
                                    relaxation_parameter,
                                    nBlocks,
                                    TPB,
                                    stream_device_to_device_1,
                                    stream_device_to_device_2,
                                    buffer_real,
                                    buffer_imag,
                                    save_data);

            auto export_thread = std::thread(&CNRectPSolver::export_single_time,
                                             this,
                                             k,
                                             buffer_real,
                                             buffer_imag,
                                             nBlocks,
                                             TPB,
                                             save_data);
            threads.push_back(std::move(export_thread));
        }
    }

    for (std::thread &th : threads)
    {
        if (th.joinable())
            th.join();
    }

    hipHostFree(h_psi_new_real);
    hipHostFree(h_psi_new_imag);
    hipHostFree(h_potential);

    if (NVTX_USE)
    {
        nvtxRangePushA("cuda free");
    }

    hipFree(d_psi_old_real);
    hipFree(d_psi_old_imag);
    hipFree(d_psi_new_real_trial);
    hipFree(d_psi_new_imag_trial);
    hipFree(d_psi_new_real);
    hipFree(d_psi_new_imag);
    hipFree(d_potential);
    hipFree(d_probability_array);
    hipFree(d_error_array);
    hipFree(d_error);
    hipFree(d_normalize_factor);

    if (NVTX_USE)
    {
        nvtxRangePop();
    }

    if (print_info)
    {
        this->domain->print_directory_info();
    }
    if (NVTX_USE)
    {
        nvtxRangePop();
    }
}

void CNRectPSolver::solve_single_time(int k,
                                      float *d_psi_old_real,
                                      float *d_psi_old_imag,
                                      float *d_psi_new_real_trial,
                                      float *d_psi_new_imag_trial,
                                      float *d_psi_new_real,
                                      float *d_psi_new_imag,
                                      float *d_potential,
                                      float *d_probability_array,
                                      float *d_normalize_factor,
                                      float *d_error_array,
                                      float *d_error,
                                      int max_iter,
                                      double tolerance,
                                      double relaxation_parameter,
                                      dim3 nBlocks,
                                      dim3 TPB,
                                      hipStream_t stream_device_to_device_1,
                                      hipStream_t stream_device_to_device_2,
                                      float *buffer_real,
                                      float *buffer_imag,
                                      bool save_data)
{
    int n_x = this->domain->get_num_grid_1();
    int n_y = this->domain->get_num_grid_2();

    float h_x = this->domain->get_infinitesimal_distance1();
    float h_y = this->domain->get_infinitesimal_distance2();

    if (k > 0)
    {
        if (NVTX_USE)
        {
            nvtxRangePushA("memcpy new to old");
        }
        hipMemcpyAsync(d_psi_old_real, d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_1);
        hipMemcpyAsync(d_psi_old_imag, d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_2);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }
    }
    if (NVTX_USE)
    {
        nvtxRangePushA((std::string("solve time ") + std::to_string(k)).c_str());
    }

    float error = 1.;
    for (auto iter = 0; iter < max_iter; ++iter)
    {
        if (error < tolerance)
        {
            break;
        }
        if (NVTX_USE)
        {
            nvtxRangePushA("memcpy new to trial");
        }
        hipMemcpyAsync(d_psi_new_real_trial, d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_1);
        hipMemcpyAsync(d_psi_new_imag_trial, d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToDevice, stream_device_to_device_2);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        if (NVTX_USE)
        {
            nvtxRangePushA("scale_prev_solution");
        }
        scale_prev_solution<<<nBlocks, TPB>>>(d_psi_new_real, d_psi_new_imag, 1 - relaxation_parameter);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        hipDeviceSynchronize();
        if (NVTX_USE)
        {
            nvtxRangePushA("cn_rect_cusolver");
        }
        cn_rect_cusolver<<<nBlocks, TPB>>>(
            d_psi_old_real,
            d_psi_old_imag,
            d_psi_new_real_trial,
            d_psi_new_imag_trial,
            d_psi_new_real,
            d_psi_new_imag,
            d_potential,
            n_x, n_y,
            this->g,
            this->domain->get_infinitesimal_distance1(),
            this->domain->get_infinitesimal_distance2(),
            this->domain->get_dt(),
            relaxation_parameter);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }
        if (NVTX_USE)
        {
            nvtxRangePushA("initialize prob and normalize factor");
        }
        hipMemset(d_probability_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
        hipMemset(d_normalize_factor, 0, sizeof(float));
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        if (NVTX_USE)
        {
            nvtxRangePushA("calculate_probability");
        }
        calculate_probability<<<nBlocks, TPB>>>(d_psi_new_real, d_psi_new_imag, d_probability_array, n_x, n_y);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        if (NVTX_USE)
        {
            nvtxRangePushA("calculate_normalize_factor");
        }
        calculate_normalize_factor<<<1, TPB.x * TPB.y>>>(d_probability_array, d_normalize_factor, TPB.x * nBlocks.x * TPB.y * nBlocks.y, h_x * h_y);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        if (NVTX_USE)
        {
            nvtxRangePushA("normalize");
        }
        normalize<<<nBlocks, TPB>>>(d_psi_new_real, d_psi_new_imag, d_normalize_factor);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        hipDeviceSynchronize();

        if (NVTX_USE)
        {
            nvtxRangePushA("initialize local and global error");
        }
        hipMemset(d_error_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
        hipMemset(d_error, 0, sizeof(float));
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        if (NVTX_USE)
        {
            nvtxRangePushA("calculate_local_error");
        }
        calculate_local_error<<<nBlocks, TPB>>>(d_psi_new_real, d_psi_new_imag, d_psi_new_real_trial, d_psi_new_imag_trial, d_error_array, n_x, n_y);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        if (NVTX_USE)
        {
            nvtxRangePushA("reduction_error");
        }
        reduction_error<<<1, TPB.x * TPB.y>>>(d_error_array, d_error, TPB.x * nBlocks.x * TPB.y * nBlocks.y);
        if (NVTX_USE)
        {
            nvtxRangePop();
        }

        hipMemcpy(&error, d_error, sizeof(float), hipMemcpyDeviceToHost);
    }
    hipMemcpyAsync(buffer_real, d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost, stream_device_to_device_1);
    hipMemcpyAsync(buffer_imag, d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost, stream_device_to_device_2);

    if (NVTX_USE)
    {
        nvtxRangePop();
    }
}

void CNRectPSolver::export_single_time(int k,
                                       float *buffer_real,
                                       float *buffer_imag,
                                       dim3 nBlocks,
                                       dim3 TPB,
                                       bool save_data)
{
    int n_x = this->domain->get_num_grid_1();
    int n_y = this->domain->get_num_grid_2();

    if (NVTX_USE)
    {
        nvtxRangePushA("save_final_data");
    }

    if (NVTX_USE)
    {
        nvtxRangePop();
    }

    if (NVTX_USE)
    {
        nvtxRangePushA("update time");
    }
    if (save_data)
    {
        std::ofstream outfile(this->domain->get_path() + std::string("Solution_") + std::to_string(k + 1) + ".txt");
        outfile << "x, y, real, imag, magn, phase " << std::endl;
        auto x_start = this->domain->get_x_start();
        auto x_end = this->domain->get_x_end();
        auto dx = this->domain->get_infinitesimal_distance1();
        auto y_start = this->domain->get_y_start();
        auto y_end = this->domain->get_y_end();
        auto dy = this->domain->get_infinitesimal_distance2();
        for (auto i = 0; i < n_x; ++i)
        {
            for (auto j = 0; j < n_y; ++j)
            {
                auto value = std::complex<float>{buffer_real[j * TPB.x * nBlocks.x + i],
                                                 buffer_imag[j * TPB.x * nBlocks.x + i]};
                float magnitude = std::abs(value);
                float phase = std::arg(value);
                outfile << x_start + i * dx << ", " << y_start + j * dy << ", ";
                outfile << buffer_real[j * TPB.x * nBlocks.x + i] << ", " << buffer_imag[j * TPB.x * nBlocks.x + i] << ", ";
                outfile << magnitude << ", " << phase;
                outfile << std::endl;
            }
        }
        outfile.close();
        this->domain->update_time();
    }
    else
        this->domain->update_time(true);
    if (NVTX_USE)
    {
        nvtxRangePop();
    }
}