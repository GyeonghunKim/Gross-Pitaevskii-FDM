#include "hip/hip_runtime.h"
#include "fe_rect_psolver.cuh"
#include <iostream>
#include <string>
#include <stdio.h>
__global__ void fe_rect_cusolver(float *psi_old_real,
                                 float *psi_old_imag,
                                 float *psi_new_real,
                                 float *psi_new_imag,
                                 float *potential,
                                 int n_x,
                                 int n_y,
                                 float g,
                                 float h_x,
                                 float h_y,
                                 float tau)
{
    __shared__ float tile_old_real[nTx][nTy];
    __shared__ float tile_old_imag[nTx][nTy];
    __shared__ float tile_potential[nTx][nTy];

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int i = (block_x + thread_x);
    int j = (block_y + thread_y);
    int striding = gridDim.x * blockDim.x;

    tile_old_real[thread_x][thread_y] = psi_old_real[j * striding + i];
    tile_old_imag[thread_x][thread_y] = psi_old_imag[j * striding + i];
    tile_potential[thread_x][thread_y] = potential[j * striding + i];

    __syncthreads();

    //  Update tile position
    psi_new_real[j * striding + i] += ((1 -
                                        tile_potential[thread_x][thread_y] -
                                        g * (tile_old_real[thread_x][thread_y] * tile_old_real[thread_x][thread_y] + tile_old_imag[thread_x][thread_y] * tile_old_imag[thread_x][thread_y])) *
                                           tile_old_real[thread_x][thread_y] +
                                       (2 * tau / (h_x * h_x) + 2 * tau / (h_y * h_y)) * tile_old_imag[thread_x][thread_y]) *
                                      (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y);
    psi_new_imag[j * striding + i] += ((1 -
                                        tile_potential[thread_x][thread_y] -
                                        g * (tile_old_real[thread_x][thread_y] * tile_old_real[thread_x][thread_y] + tile_old_imag[thread_x][thread_y] * tile_old_imag[thread_x][thread_y])) *
                                           tile_old_imag[thread_x][thread_y] -
                                       (2 * tau / (h_x * h_x) + 2 * tau / (h_y * h_y)) * tile_old_real[thread_x][thread_y]) *
                                      (i >= 0) * (i < n_x) * (j >= 0) * (j < n_y);

    // Update left position
    psi_new_real[j * striding + i - 1] += (-(tau / (h_x * h_x)) * tile_old_imag[thread_x][thread_y]) * (i > 0) * (i < n_x) * (j >= 0) * (j < n_y);
    psi_new_imag[j * striding + i - 1] += ((tau / (h_x * h_x)) * tile_old_real[thread_x][thread_y]) * (i > 0) * (i < n_x) * (j >= 0) * (j < n_y);

    // Update right position
    psi_new_real[j * striding + i + 1] += (-(tau / (h_x * h_x)) * tile_old_imag[thread_x][thread_y]) * (i >= 0) * (i < (n_x - 1)) * (j >= 0) * (j < n_y);
    psi_new_imag[j * striding + i + 1] += ((tau / (h_x * h_x)) * tile_old_real[thread_x][thread_y]) * (i >= 0) * (i < (n_x - 1)) * (j >= 0) * (j < n_y);

    // Update down position
    psi_new_real[(j - 1) * striding + i] += (-(tau / (h_y * h_y)) * tile_old_imag[thread_x][thread_y]) * (i >= 0) * (i < n_x) * (j > 0) * (j < n_y);
    psi_new_imag[(j - 1) * striding + i] += ((tau / (h_y * h_y)) * tile_old_real[thread_x][thread_y]) * (i >= 0) * (i < n_x) * (j > 0) * (j < n_y);

    // Update up position
    psi_new_real[(j + 1) * striding + i] += (-(tau / (h_y * h_y)) * tile_old_imag[thread_x][thread_y]) * (i >= 0) * (i < n_x) * (j >= 0) * (j < (n_y - 1));
    psi_new_imag[(j + 1) * striding + i] += ((tau / (h_y * h_y)) * tile_old_real[thread_x][thread_y]) * (i >= 0) * (i < n_x) * (j >= 0) * (j < (n_y - 1));
}

FERectPSolver::FERectPSolver(
    // std::function<float(float, float)> potential_,
    float g_,
    RectangularDomain *domain_)
    : BaseSolver(g_)
{
    this->domain = domain_;
    // this->generate_potential_grid();
};
// void FERectPSolver::generate_potential_grid()
// {
//     int num_grid_1 = this->domain->get_num_grid_1();
//     int num_grid_2 = this->domain->get_num_grid_2();
//     float x_start = this->domain->at(0, 0, 0)->x;
//     float y_start = this->domain->at(0, 0, 0)->y;
//     float x_end = this->domain->at(num_grid_1 - 1, num_grid_2 - 1, 0)->x;
//     float y_end = this->domain->at(num_grid_1 - 1, num_grid_2 - 1, 0)->y;
//     this->potential_grid = RectangularSpatialGrid(num_grid_1, num_grid_2, x_start, x_end, y_start, y_end);
//     for (auto i = 0; i < num_grid_1; ++i)
//     {
//         for (auto j = 0; j < num_grid_2; ++j)
//         {
//             auto point = potential_grid.at(i, j);
//             point->value = {this->potential_func(point->x, point->y), 0};
//         }
//     }
// };
float FERectPSolver::get_potential_value(int i, int j)
{
    return this->domain->potential_grid.at(i, j)->value.real();
}
/**
 * @brief Time differential of phi
 *
 * @param i index for x
 * @param j index for y
 * @param k index for time(t)
 * @return std::complex<float> time differential at x, y, t
 */
std::complex<float> FERectPSolver::temporal_equation(int i, int j, int k)
{
    // Use five stencil method
    auto point_data = this->domain->at(i, j, k);

    // l,r,d,u denotes left, right, down, up value
    // Check boundary
    auto point_data_l = this->domain->at(i - 1, j, k);
    if (i <= 0)
        point_data_l = new GridPoint(0., 0., std::complex<float>{0, 0});
    auto point_data_d = this->domain->at(i, j - 1, k);
    if (j <= 0)
        point_data_d = new GridPoint(0., 0., std::complex<float>{0, 0});
    auto point_data_r = this->domain->at(i + 1, j, k);
    if (i >= (this->domain->get_num_grid_1()) - 1)
        point_data_r = new GridPoint(0., 0., std::complex<float>{0, 0});
    auto point_data_u = this->domain->at(i, j + 1, k);
    if (j >= (this->domain->get_num_grid_2()) - 1)
        point_data_u = new GridPoint(0., 0., std::complex<float>{0, 0});

    // potential at x, y
    float V_ij = this->get_potential_value(i, j);
    // this->potential_func(point_data->x, point_data->y);

    // g * |psi(x,y)|^2
    float additional_term = (this->g) * (std::abs(point_data->value)) * (std::abs(point_data->value));

    // Set infinitesimal value
    float dx = this->domain->get_infinitesimal_distance1();
    float dy = this->domain->get_infinitesimal_distance2();
    // df denote time differential of dt (d(psi)/dt)
    //  = (laplace - V-g|psi|^2) psi
    std::complex<float> df =
        +((point_data_r->value) + (point_data_l->value) - (point_data->value) * std::complex<float>{2}) / (std::complex<float>{dx * dx}) + ((point_data_u->value) + (point_data_d->value) - (point_data->value) * std::complex<float>{2}) / (std::complex<float>{dy * dy}) - (V_ij + additional_term) * (point_data->value);
    df *= std::complex<float>{0, 1};
    return df;
};

void FERectPSolver::solve_single_time(int k)
{
    int n_x = this->domain->get_num_grid_1();
    int n_y = this->domain->get_num_grid_2();
    float dt = this->domain->get_dt();

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *h_psi_old_real = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *h_psi_old_imag = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *h_psi_new_real = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *h_psi_new_imag = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *h_potential = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *d_psi_old_real, *d_psi_old_imag, *d_psi_new_real, *d_psi_new_imag, *d_potential;
    hipMalloc((float **)&d_psi_old_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_old_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_potential, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);

    std::complex<float> wave_func;
    float potential_value;
    for (int i = 0; i < n_x; ++i)
    {
        for (int j = 0; j < n_y; ++j)
        {
            wave_func = this->domain->at(i, j, k)->value;
            potential_value = this->domain->potential_grid.at(i, j)->value.real();
            h_psi_old_real[j * TPB.x * nBlocks.x + i] = wave_func.real();
            h_psi_old_imag[j * TPB.x * nBlocks.x + i] = wave_func.imag();
            h_psi_new_real[j * TPB.x * nBlocks.x + i] = 0.;
            h_psi_new_imag[j * TPB.x * nBlocks.x + i] = 0.;
            h_potential[j * TPB.x * nBlocks.x + i] = potential_value;
        }
    }

    hipMemcpy(d_psi_new_real, h_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_psi_new_imag, h_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_psi_old_real, h_psi_old_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_psi_old_imag, h_psi_old_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_potential, h_potential, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    fe_rect_cusolver<<<nBlocks, TPB>>>(
        d_psi_old_real,
        d_psi_old_imag,
        d_psi_new_real,
        d_psi_new_imag,
        d_potential,
        n_x, n_y,
        this->g,
        this->domain->get_infinitesimal_distance1(),
        this->domain->get_infinitesimal_distance2(),
        this->domain->get_dt());
    hipDeviceSynchronize();
    hipMemcpy(h_psi_new_real, d_psi_new_real, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    hipMemcpy(h_psi_new_imag, d_psi_new_imag, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);

    for (int i = 0; i < n_x; ++i)
    {
        for (int j = 0; j < n_y; ++j)
        {
            this->domain->at(i, j, k + 1)->value = h_psi_new_real[j * TPB.x * nBlocks.x + i] + std::complex<float>{0, 1.} * h_psi_new_imag[j * TPB.x * nBlocks.x + i];
        }
    }
}

void FERectPSolver::solve(std::string dir_name)
{
    int time_length = this->domain->get_num_times();

    for (int k = 0; k < time_length - 1; ++k)
    {
        // std::cout << "Time step: " << k << std::endl;
        this->solve_single_time(k);
        this->domain->normalize(k + 1);
    }
    this->domain->generate_txt_file(std::string{"Forward_Euler_Result"} + dir_name);
}
