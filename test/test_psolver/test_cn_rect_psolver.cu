#include "hip/hip_runtime.h"
#include "test_cn_rect_psolver.cuh"
#include "../../src/solver/parallel_solver/crank_nicolson/cn_rect_psolver.cuh"
#include "../../src/potential/harmonic_potential.h"
#include "../../src/utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <functional>
#include <iostream>
#include <complex>
#include "gtest/gtest.h"

TEST(CNPSolverTest, InitializeSolveTest)
{
    bool all_passed = true;
    RectangularDomain *domain = (new RectangularDomain(32, 32, 0, 1, 3, -10, 10, -10, 10));
    auto initial_cond_function = [](float x, float y)
    { return std::complex<float>{1. * expf(-(x * x + y * y) / (1))}; };
    auto *initial_condition = new InitialCondition(initial_cond_function);
    initial_condition->assign_to_domain(domain);

    auto *potential = new HarmonicPotential(3, 5);
    potential->calcualte_potential_in_grid(domain);

    float g = -1;
    CNRectPSolver solver = CNRectPSolver(g, domain, 0);

    solver.solve(1e-11, 101);

    ASSERT_FLOAT_EQ((*domain).at(10, 10, 1)->value.real(), 0);
    ASSERT_FLOAT_EQ((*domain).at(10, 10, 1)->value.imag(), 0);
}

// #include "src/solver/parallel_solver/crank_nicolson/cn_rect_psolver.cuh"

void test_normalize(bool *passed)
{
    *passed = true;
    int n_x = 17;
    int n_y = 17;
    float h_x = 1;
    float h_y = 1;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *real_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *prob_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float normalize_factor = 0;

    for (auto i = 0; i < n_x; ++i)
    {
        for (auto j = 0; j < n_y; ++j)
        {
            real_array[TPB.x * nBlocks.x * j + i] = 1.;
            imag_array[TPB.x * nBlocks.x * j + i] = 1.;
        }
    }
    float *d_real_array, *d_imag_array, *d_prob_array, *d_normalize_factor;

    hipMalloc((float **)&d_real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_prob_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_normalize_factor, sizeof(float));
    hipMemcpy(d_real_array, real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array, imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemset(&d_prob_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemset(&d_normalize_factor, 0, sizeof(float));
    calculate_probability<<<nBlocks, TPB>>>(d_real_array, d_imag_array, d_prob_array, n_x, n_y);
    calculate_normalize_factor<<<1, TPB.x * TPB.y>>>(d_prob_array, d_normalize_factor, TPB.x * nBlocks.x * TPB.y * nBlocks.y, h_x * h_y);
    hipMemcpy(&normalize_factor, d_normalize_factor, sizeof(float), hipMemcpyDeviceToHost);

    if (!is_close(normalize_factor, sqrt(2 * n_x * n_y), 1e-12))
    {
        *passed = false;
    }
    normalize<<<nBlocks, TPB>>>(d_real_array, d_imag_array, d_normalize_factor);
    hipMemcpy(real_array, d_real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    hipMemcpy(imag_array, d_imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    if (!is_close(real_array[0], 1 / sqrt(2 * n_x * n_y), 1e-12))
    {
        *passed = false;
    }
    if (!is_close(imag_array[0], 1 / sqrt(2 * n_x * n_y), 1e-12))
    {
        *passed = false;
    }
    if (!is_close(real_array[n_x + 1], 0, 1e-12))
    {
        *passed = false;
    }
    if (!is_close(real_array[TPB.x * nBlocks.x * n_y + 1], 0, 1e-12))
    {
        *passed = false;
    }
    fileout_debug(real_array, TPB.x * nBlocks.x, TPB.y * nBlocks.y, "normalized_real.txt");
    fileout_debug(imag_array, TPB.x * nBlocks.x, TPB.y * nBlocks.y, "normalized_imag.txt");

    hipFree(d_real_array);
    hipFree(d_imag_array);
    hipFree(d_normalize_factor);
    hipFree(d_prob_array);
}

void test_error_calculation(bool *passed)
{
    *passed = true;
    int n_x = 17;
    int n_y = 17;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *real_array_1 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array_1 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *real_array_2 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array_2 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *error_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float error;
    for (auto i = 0; i < n_x; ++i)
    {
        for (auto j = 0; j < n_y; ++j)
        {
            real_array_1[TPB.x * nBlocks.x * j + i] = 1.;
            imag_array_1[TPB.x * nBlocks.x * j + i] = 1.;
            real_array_2[TPB.x * nBlocks.x * j + i] = 2.;
            imag_array_2[TPB.x * nBlocks.x * j + i] = 2.;
        }
    }
    float *d_real_array_1, *d_imag_array_1, *d_real_array_2, *d_imag_array_2, *d_error_array, *d_error;

    hipMalloc((float **)&d_real_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_real_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error, sizeof(float));

    hipMemcpy(d_real_array_1, real_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array_1, imag_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_real_array_2, real_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array_2, imag_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemset(&d_error_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemset(&d_error, 0, sizeof(float));

    calculate_local_error<<<nBlocks, TPB>>>(d_real_array_1, d_imag_array_1, d_real_array_2, d_imag_array_2, d_error_array, n_x, n_y);
    hipMemcpy(error_array, d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    fileout_debug(error_array, TPB.x * nBlocks.x, TPB.y * nBlocks.y, "error_array.txt");
    if (!is_close(error_array[0], 2, 1e-12))
    {
        *passed = false;
    }

    if (!is_close(error_array[n_x], 0, 1e-12))
    {
        *passed = false;
    }
    reduction_error<<<1, TPB.x * TPB.y>>>(d_error_array, d_error, TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemcpy(&error, d_error, sizeof(float), hipMemcpyDeviceToHost);
    if (!is_close(error, 2 * n_x * n_y, 1e-12))
    {
        *passed = false;
    }

    hipFree(d_real_array_1);
    hipFree(d_imag_array_1);
    hipFree(d_real_array_2);
    hipFree(d_imag_array_2);
    hipFree(d_error_array);
    hipFree(d_error);
}