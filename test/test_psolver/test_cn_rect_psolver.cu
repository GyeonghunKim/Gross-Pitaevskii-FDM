#include "hip/hip_runtime.h"
#include "test_cn_rect_psolver.cuh"
#include "../../src/solver/parallel_solver/crank_nicolson/cn_rect_psolver.cuh"
#include "../../src/potential/harmonic_potential.h"
#include "../../src/utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <mpi.h>
#include <functional>
#include <iostream>
#include <complex>
#include "gtest/gtest.h"

TEST(CNPSolverTest, InitializeSolveTest)
{
    MPI_Comm comm = MPI_COMM_WORLD;
    int rank, size;
    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);
    
    bool all_passed = true;
    RectangularDomain *domain = (new RectangularDomain(32, 32, 0, 1, 3, -10, 10, -10, 10));
    auto initial_cond_function = [](float x, float y)
    { return std::complex<float>{1. * expf(-(x * x + y * y) / (1))}; };
    auto *initial_condition = new InitialCondition(initial_cond_function);
    initial_condition->assign_to_domain(domain);

    auto *potential = new HarmonicPotential(3, 5);
    potential->calcualte_potential_in_grid(domain);

    float g = -1;
    CNRectPSolver solver = CNRectPSolver(g, domain, 0);

    solver.solve(1e-11, 101, std::to_string(rank), false, false);
    ASSERT_TRUE(all_passed);
}

TEST(CNPSolverTest, NormalizeTest)
{
    int n_x = 17;
    int n_y = 17;
    float h_x = 1;
    float h_y = 1;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *real_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *prob_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float normalize_factor = 0;

    for (auto i = 0; i < n_x; ++i)
    {
        for (auto j = 0; j < n_y; ++j)
        {
            real_array[TPB.x * nBlocks.x * j + i] = 1.;
            imag_array[TPB.x * nBlocks.x * j + i] = 1.;
        }
    }
    float *d_real_array, *d_imag_array, *d_prob_array, *d_normalize_factor;

    hipMalloc((float **)&d_real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_prob_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_normalize_factor, sizeof(float));
    hipMemcpy(d_real_array, real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array, imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemset(&d_prob_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemset(&d_normalize_factor, 0, sizeof(float));
    calculate_probability<<<nBlocks, TPB>>>(d_real_array, d_imag_array, d_prob_array, n_x, n_y);
    calculate_normalize_factor<<<1, TPB.x * TPB.y>>>(d_prob_array, d_normalize_factor, TPB.x * nBlocks.x * TPB.y * nBlocks.y, h_x * h_y);
    hipMemcpy(&normalize_factor, d_normalize_factor, sizeof(float), hipMemcpyDeviceToHost);

    ASSERT_FLOAT_EQ(normalize_factor, sqrt(2 * n_x * n_y));

    normalize<<<nBlocks, TPB>>>(d_real_array, d_imag_array, d_normalize_factor);
    hipMemcpy(real_array, d_real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    hipMemcpy(imag_array, d_imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);

    ASSERT_FLOAT_EQ(real_array[0], 1 / sqrt(2 * n_x * n_y));
    ASSERT_FLOAT_EQ(imag_array[0], 1 / sqrt(2 * n_x * n_y));

    hipFree(d_real_array);
    hipFree(d_imag_array);
    hipFree(d_normalize_factor);
    hipFree(d_prob_array);
}
TEST(CNPSolverTest, ErrorEstimation)
{
    int n_x = 17;
    int n_y = 17;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *real_array_1 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array_1 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *real_array_2 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array_2 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *error_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float error;
    for (auto i = 0; i < n_x; ++i)
    {
        for (auto j = 0; j < n_y; ++j)
        {
            real_array_1[TPB.x * nBlocks.x * j + i] = 1.;
            imag_array_1[TPB.x * nBlocks.x * j + i] = 1.;
            real_array_2[TPB.x * nBlocks.x * j + i] = 2.;
            imag_array_2[TPB.x * nBlocks.x * j + i] = 2.;
        }
    }
    float *d_real_array_1, *d_imag_array_1, *d_real_array_2, *d_imag_array_2, *d_error_array, *d_error;

    hipMalloc((float **)&d_real_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_real_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error, sizeof(float));

    hipMemcpy(d_real_array_1, real_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array_1, imag_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_real_array_2, real_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array_2, imag_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemset(&d_error_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemset(&d_error, 0, sizeof(float));

    calculate_local_error<<<nBlocks, TPB>>>(d_real_array_1, d_imag_array_1, d_real_array_2, d_imag_array_2, d_error_array, n_x, n_y);
    hipMemcpy(error_array, d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    ASSERT_FLOAT_EQ(error_array[0], 2);
    ASSERT_FLOAT_EQ(error_array[n_x], 0);
    reduction_error<<<1, TPB.x * TPB.y>>>(d_error_array, d_error, TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemcpy(&error, d_error, sizeof(float), hipMemcpyDeviceToHost);
    ASSERT_FLOAT_EQ(error, 2 * n_x * n_y);

    hipFree(d_real_array_1);
    hipFree(d_imag_array_1);
    hipFree(d_real_array_2);
    hipFree(d_imag_array_2);
    hipFree(d_error_array);
    hipFree(d_error);
}