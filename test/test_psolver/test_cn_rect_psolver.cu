#include "hip/hip_runtime.h"
#include "test_cn_rect_psolver.cuh"
#include "src/potential/harmonic_potential.h"

// #include "src/solver/parallel_solver/crank_nicolson/cn_rect_psolver.cuh"

void test_normalize(bool *passed)
{
    *passed = true;
    int n_x = 17;
    int n_y = 17;
    float h_x = 1;
    float h_y = 1;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *real_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *prob_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float normalize_factor = 0;

    for (auto i = 0; i < n_x; ++i)
    {
        for (auto j = 0; j < n_y; ++j)
        {
            real_array[TPB.x * nBlocks.x * j + i] = 1.;
            imag_array[TPB.x * nBlocks.x * j + i] = 1.;
        }
    }
    float *d_real_array, *d_imag_array, *d_prob_array, *d_normalize_factor;

    hipMalloc((float **)&d_real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_prob_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_normalize_factor, sizeof(float));
    hipMemcpy(d_real_array, real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array, imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemset(&d_prob_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemset(&d_normalize_factor, 0, sizeof(float));
    calculate_probability<<<nBlocks, TPB>>>(d_real_array, d_imag_array, d_prob_array, n_x, n_y);
    calculate_normalize_factor<<<1, TPB.x * TPB.y>>>(d_prob_array, d_normalize_factor, TPB.x * nBlocks.x * TPB.y * nBlocks.y, h_x * h_y);
    hipMemcpy(&normalize_factor, d_normalize_factor, sizeof(float), hipMemcpyDeviceToHost);

    if (!is_close(normalize_factor, sqrt(2 * n_x * n_y), 1e-12))
    {
        *passed = false;
    }
    normalize<<<nBlocks, TPB>>>(d_real_array, d_imag_array, d_normalize_factor);
    hipMemcpy(real_array, d_real_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    hipMemcpy(imag_array, d_imag_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    if (!is_close(real_array[0], 1 / sqrt(2 * n_x * n_y), 1e-12))
    {
        *passed = false;
    }
    if (!is_close(imag_array[0], 1 / sqrt(2 * n_x * n_y), 1e-12))
    {
        *passed = false;
    }
    if (!is_close(real_array[n_x + 1], 0, 1e-12))
    {
        *passed = false;
    }
    if (!is_close(real_array[TPB.x * nBlocks.x * n_y + 1], 0, 1e-12))
    {
        *passed = false;
    }
    fileout_debug(real_array, TPB.x * nBlocks.x, TPB.y * nBlocks.y, "normalized_real.txt");
    fileout_debug(imag_array, TPB.x * nBlocks.x, TPB.y * nBlocks.y, "normalized_imag.txt");

    hipFree(d_real_array);
    hipFree(d_imag_array);
    hipFree(d_normalize_factor);
    hipFree(d_prob_array);
}

void test_error_calculation(bool *passed)
{
    *passed = true;
    int n_x = 17;
    int n_y = 17;

    dim3 TPB(nTx, nTy);
    dim3 nBlocks(n_x / nTx + (n_x % nTx != 0), n_y / nTy + (n_y % nTy != 0));
    float *real_array_1 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array_1 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *real_array_2 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *imag_array_2 = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float *error_array = (float *)malloc(sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    float error;
    for (auto i = 0; i < n_x; ++i)
    {
        for (auto j = 0; j < n_y; ++j)
        {
            real_array_1[TPB.x * nBlocks.x * j + i] = 1.;
            imag_array_1[TPB.x * nBlocks.x * j + i] = 1.;
            real_array_2[TPB.x * nBlocks.x * j + i] = 2.;
            imag_array_2[TPB.x * nBlocks.x * j + i] = 2.;
        }
    }
    float *d_real_array_1, *d_imag_array_1, *d_real_array_2, *d_imag_array_2, *d_error_array, *d_error;

    hipMalloc((float **)&d_real_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_real_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_imag_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMalloc((float **)&d_error, sizeof(float));

    hipMemcpy(d_real_array_1, real_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array_1, imag_array_1, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_real_array_2, real_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemcpy(d_imag_array_2, imag_array_2, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyHostToDevice);
    hipMemset(&d_error_array, 0, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemset(&d_error, 0, sizeof(float));

    calculate_local_error<<<nBlocks, TPB>>>(d_real_array_1, d_imag_array_1, d_real_array_2, d_imag_array_2, d_error_array, n_x, n_y);
    hipMemcpy(error_array, d_error_array, sizeof(float) * TPB.x * nBlocks.x * TPB.y * nBlocks.y, hipMemcpyDeviceToHost);
    fileout_debug(error_array, TPB.x * nBlocks.x, TPB.y * nBlocks.y, "error_array.txt");
    if (!is_close(error_array[0], 2, 1e-12))
    {
        *passed = false;
    }

    if (!is_close(error_array[n_x], 0, 1e-12))
    {
        *passed = false;
    }
    reduction_error<<<1, TPB.x * TPB.y>>>(d_error_array, d_error, TPB.x * nBlocks.x * TPB.y * nBlocks.y);
    hipMemcpy(&error, d_error, sizeof(float), hipMemcpyDeviceToHost);
    if (!is_close(error, 2 * n_x * n_y, 1e-12))
    {
        *passed = false;
    }

    hipFree(d_real_array_1);
    hipFree(d_imag_array_1);
    hipFree(d_real_array_2);
    hipFree(d_imag_array_2);
    hipFree(d_error_array);
    hipFree(d_error);
}