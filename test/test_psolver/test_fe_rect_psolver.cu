#include "../../src/solver/base_solver.h"
#include "../../src/solver/parallel_solver/forward_euler/fe_rect_psolver.cuh"

#include "../../src/potential/harmonic_potential.h"
#include "../../src/utils.h"
#include <iostream>
#include <complex>

#include "gtest/gtest.h"

TEST(FEPSolverTest, InitializeSolveTest)
{
    bool all_passed = true;
    // std::function<float(float, float)> potential;

    float g;
    RectangularDomain *domain = (new RectangularDomain(32, 32, 0, 1e-4, 2, -5, 5, -5, 5));
    auto initial_cond_function = [](float x, float y)
    { return std::complex<float>{1e-10}; };

    auto *initial_condition = new InitialCondition(initial_cond_function);
    initial_condition->assign_to_domain(domain);

    auto *potential = new HarmonicPotential(3, 5);
    potential->calcualte_potential_in_grid(domain);
    g = -1.;
    FERectPSolver solver = FERectPSolver(g, domain, 0);

    solver.solve("", false, false);

    ASSERT_FLOAT_EQ((*domain).at(10, 10, 1)->value.real(), 0.096875511);
}
